#include "hip/hip_runtime.h"
#include <algorithm>
#include <cstdlib>
#include <iostream>
#include <limits>
#include <random>
#include <vector>
#include <float.h>
#include <chrono>
#define NUMBER_OF_CLUSTERS 8

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

struct Datum {
    float x{0};
    float y{0};
    float z{0};
};

using Points = std::vector<Datum>;

float square(float a) {
    return a*a;
}

float squared_distance(Datum a, Datum b) {
    return square(a.x - b.x) + square(a.y - b.y) + square(a.z - b.z);
}

Points kmeansCPU(const Points& points, Points centroids, size_t number_of_examples, float threshold) {
    std::vector<size_t> assignments(number_of_examples);
    float changed = number_of_examples;
    while(changed/number_of_examples > threshold){
        //printf("changed is %f\n", changed);
        changed = 0;
        for(int example = 0; example < number_of_examples - 1; ++example) {
            float currentDistance = std::numeric_limits<float>::max();
            size_t currentCentroid = 0;
            for(int centroid = 0; centroid < NUMBER_OF_CLUSTERS - 1; ++centroid) {
                if(squared_distance(points[example], centroids[centroid]) < currentDistance){
                    currentDistance = squared_distance(points[example], centroids[centroid]);
                    currentCentroid = centroid;
                }
            }
            if(assignments[example] != currentCentroid) ++changed;
            assignments[example] = currentCentroid;
        }
        std::vector<size_t> counter(NUMBER_OF_CLUSTERS, 0);
        Points new_centroids(NUMBER_OF_CLUSTERS);
        for(int assignment = 0; assignment < assignments.size() - 1; ++assignment) {
            new_centroids[assignments[assignment]].x += points[assignment].x;
            new_centroids[assignments[assignment]].y += points[assignment].y;
            new_centroids[assignments[assignment]].z += points[assignment].z;
            counter[assignments[assignment]] = counter[assignments[assignment]] + 1;
        }
        for(int centroid = 0; centroid < NUMBER_OF_CLUSTERS - 1; ++centroid) {
            const auto count = std::max<size_t>(1, counter[centroid]);
            centroids[centroid].x = new_centroids[centroid].x/count;
            centroids[centroid].y = new_centroids[centroid].y/count;
            centroids[centroid].z = new_centroids[centroid].z/count;
        }
        
    }
    return centroids;
    }

void runCPU(Points points, Points centroids, size_t number_of_examples, float threshold)
{
    printf("Starting sequential kmeans\n");
    auto start = std::chrono::system_clock::now();
    Points result = kmeansCPU(points, centroids, number_of_examples, threshold);
    auto end = std::chrono::system_clock::now();
    printf("\n");
    for (auto i: result)
        std::cout << i.x << ' ' << i.y << ' ' << i.z << "\n";
    printf("\n");

    float duration = 1000.0*std::chrono::duration<float>(end - start).count();
    printf("\nElapsed time in milliseconds : %f ms.\n\n", duration);
    
}

__device__ float distance_squared(float x1, float x2, float y1, float y2, float z1, float z2) {
    return (x1-x2)*(x1-x2) + (y1-y2)*(y1-y2) + (z1-z2)*(z1-z2);
}

__global__ void distances_calculation(Datum* d_points, Datum* d_centroids, Datum* new_centroids, size_t* counters, size_t* assignments, size_t number_of_examples, size_t number_of_clusters, size_t* if_changed) {
    size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid >= number_of_examples) return;
    size_t local_tid = blockIdx.x;
    extern __shared__ Datum local_centroids[];
    //coalesced read
    float _distance;
    float _x = d_points[tid].x;
    float _y = d_points[tid].y;
    float _z = d_points[tid].z;

    float currentDistance = FLT_MAX;
    size_t currentCentroid = 0;
    if(local_tid < number_of_clusters) {
        local_centroids[tid]= d_centroids[tid];
    }
    for(int i = 0; i < number_of_clusters; ++i) {
        _distance = distance_squared(_x, local_centroids[i].x, _y,local_centroids[i].y , _z, local_centroids[i].z);
        if(_distance < currentDistance) {
            currentCentroid = i;
            currentDistance = _distance;
        }
    }

    if_changed[tid] = 0;
    if(assignments[tid] != currentCentroid) {
        if_changed[tid] = 1;
        assignments[tid] = currentCentroid;
    }

    printf("im tid %d\n", tid);

      // Slow but simple.
    atomicAdd(&new_centroids[currentCentroid].x, _x);
    atomicAdd(&new_centroids[currentCentroid].y, _y);
    atomicAdd(&new_centroids[currentCentroid].z, _z);
    atomicAdd(&counters[currentCentroid], (size_t)1);

}

void runGPU(Points points, Points centroids, size_t number_of_examples, float threshold, size_t number_of_clusters){
    //TODO initialization and CUDAMallocs
    float changed = number_of_examples;
    Datum* d_points;
    size_t* if_changed;
    Datum* d_centroids;
    Datum* new_centroids;
    size_t* counters;
    size_t* assignments;
    //we will be accessing memory structures concurrently -> AoS makes more sense than SoA
    hipMallocManaged(&if_changed, points.size()*sizeof(size_t));
    hipMallocManaged(&d_points, points.size()*sizeof(Datum));
    hipMallocManaged(&d_centroids, centroids.size()*sizeof(Datum));
    hipMallocManaged(&new_centroids, centroids.size()*sizeof(Datum));
    hipMallocManaged(&counters, centroids.size()*sizeof(size_t));
    hipMallocManaged(&assignments, points.size()*sizeof(size_t));
    for(int i = 0; i < number_of_examples; ++i) {
        d_points[i] = points[i];
    }
    for(int i = 0; i < number_of_clusters; ++i) {
        d_centroids[i] = centroids[i];
        new_centroids[i].x = 0;
        new_centroids[i].y = 0;
        new_centroids[i].z = 0;
    }
    
    int num_threads = 1024;
    int num_blocks = (number_of_examples + num_threads - 1) / num_threads;
    int mem = number_of_clusters*sizeof(Datum);
    //while(changed/number_of_examples > threshold) {
        changed = 0;
        distances_calculation<<<num_threads, num_blocks, mem>>>(d_points, d_centroids, new_centroids, counters, assignments, number_of_examples, number_of_clusters, if_changed);
        gpuErrchk( hipPeekAtLastError() );
        gpuErrchk( hipDeviceSynchronize() );
        // move_centroids<<<1, number_of_clusters>>>>();
        // gpuErrchk( hipPeekAtLastError() );
        // gpuErrchk( hipDeviceSynchronize() );

    //}

    //TODO hipFree
    hipFree(d_points);
    hipFree(d_centroids);
    hipFree(new_centroids);
    hipFree(counters);

}

int main(int argc, char *argv[])
{
    if(argc < 2)
    { 
        printf("Not enough arguments\n 1st argument -> number of examples to generate divisible by 8\n 2nd argument -> maximal absolute value on grid \n 3rd argument -> 0-1 threshold for stopping iterating\n\n");
        return 0;
    }
    //default number of clusters = 8;
    size_t number_of_examples = atoi(argv[1]);
    float grid_max_value = atof(argv[2]);
    float threshold = atof(argv[3]);
    size_t number_of_clusters = NUMBER_OF_CLUSTERS;
    if(number_of_examples%number_of_clusters != 0) {
        printf("The number of examples has to be divisible by 8\n\n");
        return 0;
    }
    Points points(number_of_examples);
    static std::random_device seed;
    static std::mt19937 random_number_generator(seed());
    std::uniform_real_distribution<float> indices_upper(grid_max_value*0.5, grid_max_value);
    std::uniform_real_distribution<float> indices_lower(-grid_max_value, -grid_max_value*0.5);

    for(int i = 0; i < number_of_examples; ++i) {
        if(i < number_of_examples / number_of_clusters){
        points[i].x = indices_lower(random_number_generator);
        points[i].y = indices_upper(random_number_generator);
        points[i].z = indices_upper(random_number_generator);
        } else if(i < 2*number_of_examples/number_of_clusters) {
        points[i].x = indices_lower(random_number_generator);
        points[i].y = indices_upper(random_number_generator);
        points[i].z = indices_lower(random_number_generator);
        } else if(i < 3*number_of_examples/number_of_clusters) {
        points[i].x = indices_upper(random_number_generator);
        points[i].y = indices_upper(random_number_generator);
        points[i].z = indices_lower(random_number_generator);
        } else if(i < 4*number_of_examples/number_of_clusters) {
        points[i].x = indices_upper(random_number_generator);
        points[i].y = indices_upper(random_number_generator);
        points[i].z = indices_upper(random_number_generator);
        } else if(i < 5*number_of_examples/number_of_clusters) {
        points[i].x = indices_upper(random_number_generator);
        points[i].y = indices_lower(random_number_generator);
        points[i].z = indices_upper(random_number_generator);
        } else if(i < 6*number_of_examples/number_of_clusters) {
        points[i].x = indices_upper(random_number_generator);
        points[i].y = indices_lower(random_number_generator);
        points[i].z = indices_lower(random_number_generator);
        } else if(i < 7*number_of_examples/number_of_clusters) {
        points[i].x = indices_lower(random_number_generator);
        points[i].y = indices_lower(random_number_generator);
        points[i].z = indices_lower(random_number_generator);
        } else if(i < number_of_examples) {
        points[i].x = indices_lower(random_number_generator);
        points[i].y = indices_lower(random_number_generator);
        points[i].z = indices_upper(random_number_generator);
        }
    }
    Points centroids(number_of_clusters);
    std::uniform_real_distribution<float> indices(0, number_of_examples - 1);
    for(auto& centroid : centroids) {
        centroid = points[indices(random_number_generator)];
    }
    // Datum PRINTING
    // for(auto& Datum : points) {
    //     printf("x is %f y is %f and z is %f \n", Datum.x, Datum.y, Datum.z);
    // }
    
    runCPU(points, centroids, number_of_examples, threshold);
    runGPU(points, centroids, number_of_examples, threshold, number_of_clusters);

    return 0;
}