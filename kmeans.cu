
#include <hip/hip_runtime.h>
#include <algorithm>
#include <cstdlib>
#include <iostream>
#include <limits>
#include <random>
#include <vector>
#include <float.h>
#include <chrono>

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

struct Datum {
    float x{0};
    float y{0};
    float z{0};
};

using Points = std::vector<Datum>;

float square(float a) {
    return a*a;
}

float squared_distance(Datum a, Datum b) {
    return square(a.x - b.x) + square(a.y - b.y) + square(a.z - b.z);
}

Points kmeansCPU(const Points& points, Points centroids, int number_of_examples, int iterations, int number_of_clusters) {
    std::vector<int> assignments(number_of_examples);
    for(int i = 0; i < iterations; ++i){
        for(int example = 0; example < number_of_examples; ++example) {
            float currentDistance = FLT_MAX;
            int currentCentroid = 0;
            for(int centroid = 0; centroid < number_of_clusters; ++centroid) {
                if(squared_distance(points[example], centroids[centroid]) < currentDistance){
                    currentDistance = squared_distance(points[example], centroids[centroid]);
                    currentCentroid = centroid;
                }
            }
            assignments[example] = currentCentroid;
        }
        // for(auto i:assignments) {
        //     printf(" %d ", i);
        // }
        // printf("\n");
        std::vector<int> counter(number_of_clusters, 0);
        Points new_centroids(number_of_clusters);
        for(int assignment = 0; assignment < assignments.size(); ++assignment) {
            new_centroids[assignments[assignment]].x += points[assignment].x;
            new_centroids[assignments[assignment]].y += points[assignment].y;
            new_centroids[assignments[assignment]].z += points[assignment].z;
            counter[assignments[assignment]] = counter[assignments[assignment]] + 1;
        }
        for(int centroid = 0; centroid < number_of_clusters; ++centroid) {
            const auto count = std::max<int>(1, counter[centroid]);
            centroids[centroid].x = new_centroids[centroid].x/count;
            centroids[centroid].y = new_centroids[centroid].y/count;
            centroids[centroid].z = new_centroids[centroid].z/count;
        }
        
    }
    return centroids;
    }

void runCPU(Points points, Points centroids, int number_of_examples, int iterations, int number_of_clusters)
{
    printf("\nStarting sequential kmeans\n");
    auto start = std::chrono::system_clock::now();
    Points result = kmeansCPU(points, centroids, number_of_examples, iterations, number_of_clusters);
    auto end = std::chrono::system_clock::now();
    printf("\n");
    for (int i = 0; i < number_of_clusters; i++){
        printf("%f  %f  %f", result[i].x, result[i].y, result[i].z);     printf("\n");}


    float duration = 1000.0*std::chrono::duration<float>(end - start).count();
    printf("\nElapsed time in milliseconds : %f ms.\n\n", duration);
    
}

__device__ float distance_squared(float x1, float x2, float y1, float y2, float z1, float z2) {
    return (x1-x2)*(x1-x2) + (y1-y2)*(y1-y2) + (z1-z2)*(z1-z2);
}
__global__ void move_centroids(float* d_centroids_x, float* d_centroids_y, float* d_centroids_z, float* d_new_centroids_x, float* d_new_centroids_y, float* d_new_centroids_z, int* counters, int number_of_clusters) 
{
    int tid = threadIdx.x;
    const int count = max(1, counters[tid]);
    d_centroids_x[tid] = d_new_centroids_x[tid]/count;
    d_centroids_y[tid] = d_new_centroids_y[tid]/count;
    d_centroids_z[tid] = d_new_centroids_z[tid]/count;
    d_new_centroids_x[tid] = 0;
    d_new_centroids_y[tid] = 0;
    d_new_centroids_z[tid] = 0;
}

__global__ void distances_calculation(float* d_points_x, float* d_points_y, float* d_points_z, float* d_centroids_x, float* d_centroids_y, float* d_centroids_z, float* d_new_centroids_x, float* d_new_centroids_y, float* d_new_centroids_z, int* counters, int number_of_examples, int number_of_clusters) 
{
    extern __shared__ float local_centroids[];
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int local_tid = threadIdx.x;
    if(tid >= number_of_examples) return;
    int currentCentroid = 0;
    //coalesced read
    float _x = d_points_x[tid];
    float _y = d_points_y[tid];
    float _z = d_points_z[tid];
    float currentDistance = FLT_MAX;

    if(local_tid < number_of_clusters) {
        local_centroids[local_tid]= d_centroids_x[local_tid];
        local_centroids[local_tid + number_of_clusters]= d_centroids_y[local_tid];
        local_centroids[local_tid + number_of_clusters + number_of_clusters]= d_centroids_z[local_tid];
    }
    __syncthreads();
    for(int i = 0; i < number_of_clusters; ++i) {
        const float _distance = distance_squared(_x, local_centroids[i], _y,local_centroids[i + number_of_clusters] , _z, local_centroids[i + 2*number_of_clusters]);
        if(_distance < currentDistance) {
            currentCentroid = i;
            currentDistance = _distance;
        }
    }

    //Slow but simple.
    //printf("tid: %d im adding to %d values %f %f %f, number of clusters is %d\n", tid, currentCentroid, _x, _y, _z, number_of_clusters);
    atomicAdd(&d_new_centroids_x[currentCentroid], _x);
    atomicAdd(&d_new_centroids_y[currentCentroid], _y);
    atomicAdd(&d_new_centroids_z[currentCentroid], _z);
    atomicAdd(&counters[currentCentroid], 1);

}

void runGPU(Points points, Points centroids, int number_of_examples, int iterations, int number_of_clusters)
{
    //TODO initialization and CUDAMallocs
    float* d_points_x;
    float* d_points_y;
    float* d_points_z;
    float* d_centroids_x;
    float* d_centroids_y;
    float* d_centroids_z;  
    float* d_new_centroids_x;
    float* d_new_centroids_y;
    float* d_new_centroids_z;
    int* counters;
    //we will be accessing memory structures concurrently -> AoS makes more sense than SoA
    hipMallocManaged(&d_points_x, points.size()*sizeof(float));
    hipMallocManaged(&d_points_y, points.size()*sizeof(float));
    hipMallocManaged(&d_points_z, points.size()*sizeof(float));
    hipMallocManaged(&d_centroids_x, centroids.size()*sizeof(float));
    hipMallocManaged(&d_centroids_y, centroids.size()*sizeof(float));
    hipMallocManaged(&d_centroids_z, centroids.size()*sizeof(float));
    hipMallocManaged(&d_new_centroids_x, centroids.size()*sizeof(float));
    hipMallocManaged(&d_new_centroids_y, centroids.size()*sizeof(float));
    hipMallocManaged(&d_new_centroids_z, centroids.size()*sizeof(float));
    hipMallocManaged(&counters, centroids.size()*sizeof(int));
    for(int i = 0; i < number_of_examples; ++i) {
        d_points_x[i] = points[i].x;
        d_points_y[i] = points[i].y;
        d_points_z[i] = points[i].z;
    }
    for(int i = 0; i < number_of_clusters; ++i) {
        d_centroids_x[i] = centroids[i].x;
        d_centroids_y[i] = centroids[i].y;
        d_centroids_z[i] = centroids[i].z;
        d_new_centroids_x[i] = 0;
        d_new_centroids_y[i] = 0;
        d_new_centroids_z[i] = 0;
    }
    
    int num_threads = 1024;
    int num_blocks = (number_of_examples + num_threads - 1) / num_threads;
    int mem = 3*number_of_clusters*sizeof(float);
    printf("Starting parallel kmeans\n");
    auto start = std::chrono::system_clock::now();
    for(int i = 0; i < iterations; ++i) {
        hipMemset(counters, 0, number_of_clusters*sizeof(int));
        distances_calculation<<<num_blocks, num_threads, mem>>>(d_points_x, d_points_y, d_points_z, d_centroids_x, d_centroids_y, d_centroids_z, d_new_centroids_x, d_new_centroids_y, d_new_centroids_z, counters, number_of_examples, number_of_clusters);
        gpuErrchk( hipPeekAtLastError() );
        gpuErrchk( hipDeviceSynchronize() );
        //for(int i = 0; i < number_of_clusters; ++i) printf("centroid sums: %f %f %f\n", d_new_centroids_x[i], d_new_centroids_y[i], d_new_centroids_z[i]);
        move_centroids<<<1, number_of_clusters>>>(d_centroids_x, d_centroids_y, d_centroids_z, d_new_centroids_x, d_new_centroids_y, d_new_centroids_z, counters, number_of_clusters);
        gpuErrchk( hipPeekAtLastError() );
        gpuErrchk( hipDeviceSynchronize() );

    }
    auto end = std::chrono::system_clock::now();
    float duration = 1000.0*std::chrono::duration<float>(end - start).count();
    printf("\nElapsed time in milliseconds : %f ms.\n\n", duration);

    for (int i = 0; i < number_of_clusters; i++){
        printf("%f  %f  %f", d_centroids_x[i], d_centroids_y[i], d_centroids_z[i]);     printf("\n");}


    hipFree(d_points_x);
    hipFree(d_points_y);
    hipFree(d_points_z);
    hipFree(d_centroids_x);
    hipFree(d_centroids_y);
    hipFree(d_centroids_z);  
    hipFree(d_new_centroids_x);
    hipFree(d_new_centroids_y);
    hipFree(d_new_centroids_z);
    hipFree(counters);

}

int main(int argc, char *argv[])
{
    if(argc < 5)
    { 
        printf("Not enough arguments\n 1st argument -> number of examples to generate divisible by 8\n 2nd argument -> maximal absolute value on grid \n 3rd argument -> number of iterations \n 4th argument -> number of clusters\n\n");
        return 0;
    }
    //default number of clusters = 8;
    int number_of_examples = atoi(argv[1]);
    float grid_max_value = atof(argv[2]);
    int iterations = atoi(argv[3]);
    int number_of_clusters = atoi(argv[4]);
    if(number_of_examples < number_of_clusters != 0) {
        printf("The number of examples has to be smaller than number of clusters\n\n");
        return 0;
    }
    Points points(number_of_examples);
    static std::random_device seed;
    static std::mt19937 random_number_generator(seed());

    {
    //8 domain generation
        // std::uniform_real_distribution<float> indices_upper(grid_max_value*0.5, grid_max_value);
        // std::uniform_real_distribution<float> indices_lower(-grid_max_value, -grid_max_value*0.5);
        // for(int i = 0; i < number_of_examples; ++i) {
        //     if(i < number_of_examples / number_of_clusters){
        //     points[i].x = indices_lower(random_number_generator);
        //     points[i].y = indices_upper(random_number_generator);
        //     points[i].z = indices_upper(random_number_generator);
        //     } else if(i < 2*number_of_examples/number_of_clusters) {
        //     points[i].x = indices_lower(random_number_generator);
        //     points[i].y = indices_upper(random_number_generator);
        //     points[i].z = indices_lower(random_number_generator);
        //     } else if(i < 3*number_of_examples/number_of_clusters) {
        //     points[i].x = indices_upper(random_number_generator);
        //     points[i].y = indices_upper(random_number_generator);
        //     points[i].z = indices_lower(random_number_generator);
        //     } else if(i < 4*number_of_examples/number_of_clusters) {
        //     points[i].x = indices_upper(random_number_generator);
        //     points[i].y = indices_upper(random_number_generator);
        //     points[i].z = indices_upper(random_number_generator);
        //     } else if(i < 5*number_of_examples/number_of_clusters) {
        //     points[i].x = indices_upper(random_number_generator);
        //     points[i].y = indices_lower(random_number_generator);
        //     points[i].z = indices_upper(random_number_generator);
        //     } else if(i < 6*number_of_examples/number_of_clusters) {
        //     points[i].x = indices_upper(random_number_generator);
        //     points[i].y = indices_lower(random_number_generator);
        //     points[i].z = indices_lower(random_number_generator);
        //     } else if(i < 7*number_of_examples/number_of_clusters) {
        //     points[i].x = indices_lower(random_number_generator);
        //     points[i].y = indices_lower(random_number_generator);
        //     points[i].z = indices_lower(random_number_generator);
        //     } else if(i < number_of_examples) {
        //     points[i].x = indices_lower(random_number_generator);
        //     points[i].y = indices_lower(random_number_generator);
        //     points[i].z = indices_upper(random_number_generator);
        //     }
        // }
    }
    std::uniform_real_distribution<float> indices_general(-grid_max_value, grid_max_value);
    for(int i = 0; i < number_of_examples; ++i) {
         points[i].x = indices_general(random_number_generator);
         points[i].y = indices_general(random_number_generator);
         points[i].z = indices_general(random_number_generator);
    }

    Points centroids(number_of_clusters);
    std::uniform_real_distribution<float> indices(0, number_of_examples - 1);
    for(auto& centroid : centroids) {
        centroid = points[indices(random_number_generator)];
    }
    //Datum PRINTING
    // for(auto& Datum : points) {
    //     printf("x is %f y is %f and z is %f \n", Datum.x, Datum.y, Datum.z);
    // }
    
    runGPU(points, centroids, number_of_examples, iterations, number_of_clusters);
    runCPU(points, centroids, number_of_examples, iterations, number_of_clusters);

    return 0;
}